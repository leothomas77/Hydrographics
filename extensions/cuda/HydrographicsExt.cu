#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <vector>
#include <limits>
#include <algorithm>

#include "../../core/core.h"
#include "../../core/maths.h"

#define CudaCheck(x) { hipError_t err = x; if (err != hipSuccess) { printf("Cuda error: %d in %s at %s:%d\n", err, #x, __FILE__, __LINE__); assert(0); } }

static const int kNumThreadsPerBlock = 256;



